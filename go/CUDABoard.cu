#include "hip/hip_runtime.h"
#include "CUDABoard.h"


namespace{
  __global__
  void initBoard(BoardPoint *boardDevice, hiprandState *state, long randSeed){
   
    int index = threadIdx.y * boardSize + threadIdx.x;

    hiprand_init(randSeed, index, 0, &state[index]);
  
    if (threadIdx.x == 0 || threadIdx.x == boardSize-1 || threadIdx.y == 0 || threadIdx.y == boardSize-1){
      boardDevice[index].color = 3;
    } else {
      boardDevice[index].color = 0;
    }
  
    //all the initial group ID will be zero..
  
  }
  __device__
  inline int invertColor(int color){
    if (color == GO_BLACK){
      return GO_WHITE;
    }else if(color == GO_WHITE){
      return GO_BLACK;
    }
    return GO_EMPTY;
  }
 

  __device__
  inline int generateRandomValue(int index, hiprandState *state){
    return hiprand(&state[index])>>3; // move left by 3 bit to make sure that it will not be negative after assigned to int.
  }
  
  __device__
  inline void updateStatus(BoardPoint *boardDevice, 
                          int index, 
                          GoColor color, 
                          int *globalLiberty, 
                          int *globalMoveValue, 
                          hiprandState *state){
     if (boardDevice[index].color == GO_EMPTY){
      // updating liberty for each point 
      atomicAdd(&globalLiberty[boardDevice[index-1].groupID], 1);
   
      if (boardDevice[index+boardSize].groupID != boardDevice[index-1].groupID){
        atomicAdd(&globalLiberty[boardDevice[index+boardSize].groupID], 1);

      } 
  
      if (boardDevice[index+1].groupID != boardDevice[index-1].groupID &&
          boardDevice[index+1].groupID != boardDevice[index+boardSize].groupID){
        atomicAdd(&globalLiberty[boardDevice[index+1].groupID], 1);

       } 
  
      if (boardDevice[index-boardSize].groupID != boardDevice[index-1].groupID &&
          boardDevice[index-boardSize].groupID != boardDevice[index+1].groupID &&
          boardDevice[index-boardSize].groupID != boardDevice[index+boardSize].groupID){
        atomicAdd(&globalLiberty[boardDevice[index-boardSize].groupID], 1);

       } 

     }
    
    __syncthreads();
    __threadfence_block();
  
    int libertyNumber = globalLiberty[boardDevice[index].groupID];
    boardDevice[index].libertyNumber = libertyNumber;
     
    __syncthreads();
    __threadfence_block();

    // computing move value for each point

    if (boardDevice[index].color == GO_EMPTY){
      if (color == GO_WHITE){
        //assuming that next move will be black, as current move is white.
        if( (boardDevice[index - 1].color == GO_WHITE && boardDevice[index-1].libertyNumber == 1) ||
            (boardDevice[index + 1].color == GO_WHITE && boardDevice[index+1].libertyNumber == 1) ||
            (boardDevice[index - boardSize].color == GO_WHITE && boardDevice[index-boardSize].libertyNumber == 1) ||
            (boardDevice[index + boardSize].color == GO_WHITE && boardDevice[index+boardSize].libertyNumber == 1)){
          globalMoveValue[index] = generateRandomValue(index, state);
        }else {
          if (boardDevice[index - 1].color == GO_EMPTY ||
              (boardDevice[index - 1].color == GO_BLACK && boardDevice[index-1].libertyNumber > 1)||
              boardDevice[index + 1].color == GO_EMPTY ||
              (boardDevice[index + 1].color == GO_BLACK && boardDevice[index+1].libertyNumber > 1)||
              boardDevice[index - boardSize].color == GO_EMPTY ||
              (boardDevice[index - boardSize].color == GO_BLACK && boardDevice[index-boardSize].libertyNumber > 1)||
              boardDevice[index + boardSize].color == GO_EMPTY ||
              (boardDevice[index + boardSize].color == GO_BLACK && boardDevice[index+boardSize].libertyNumber > 1)){
            globalMoveValue[index] = generateRandomValue(index, state);
          }else{
            globalMoveValue[index] = -1;
          }
        }
      }else if (color == GO_BLACK){
        //assuming that next move will be white, as current move is black.
        if( (boardDevice[index - 1].color == GO_BLACK && boardDevice[index-1].libertyNumber == 1) ||
            (boardDevice[index + 1].color == GO_BLACK && boardDevice[index+1].libertyNumber == 1) ||
            (boardDevice[index - boardSize].color == GO_BLACK && boardDevice[index-boardSize].libertyNumber == 1) ||
            (boardDevice[index + boardSize].color == GO_BLACK && boardDevice[index+boardSize].libertyNumber == 1)){
          globalMoveValue[index] = generateRandomValue(index, state);
        }else {
         if (boardDevice[index - 1].color == GO_EMPTY ||
            (boardDevice[index - 1].color == GO_WHITE && boardDevice[index-1].libertyNumber > 1)||
            boardDevice[index + 1].color == GO_EMPTY ||
            (boardDevice[index + 1].color == GO_WHITE && boardDevice[index+1].libertyNumber > 1)||
            boardDevice[index - boardSize].color == GO_EMPTY ||
            (boardDevice[index - boardSize].color == GO_WHITE && boardDevice[index-boardSize].libertyNumber > 1)||
            boardDevice[index + boardSize].color == GO_EMPTY ||
            (boardDevice[index + boardSize].color == GO_WHITE && boardDevice[index+boardSize].libertyNumber > 1)){
            globalMoveValue[index] = generateRandomValue(index, state);
          }else{
            globalMoveValue[index] = -1;
          }
        }
      }
    }else{
       // current point is not empty, it is ilegal move, set the value to zero.
       globalMoveValue[index] = -1;
     }

    __syncthreads();
    __threadfence_block();
 
   boardDevice[index].moveValue = globalMoveValue[index];
 
  }

  __device__
  void playStone(BoardPoint *boardDevice, 
                DebugFlag *debugFlagDevice, 
                int *selectedMove, 
                GoColor color, 
                int *globalLiberty, 
                int *globalMoveValue, 
                hiprandState *state){
    int index = threadIdx.y*boardSize + threadIdx.x;
    int playPoint = *selectedMove;
    GoColor enemyColor = invertColor(color);

    __shared__ int targetGroupID[4];
    __shared__ int removedGroupID[4];
    //__shared__ bool hasStoneRemoved;
  
  
    if (threadIdx.y == 0 || threadIdx.y == boardSize || threadIdx.x == 0 || threadIdx.x == boardSize){
      // out of the real board, reset the liberty of Group 0 to 0, then return.
      globalLiberty[0] = 0;
      return;
    }
  
  
    if (index == playPoint){
        boardDevice[index].color = color;
        boardDevice[index].groupID = index;
  
        if (boardDevice[index+1].color == color){
          targetGroupID[0] = boardDevice[index+1].groupID;
        }else if(boardDevice[index + 1].color == enemyColor){
          if (boardDevice[index + 1].libertyNumber == 1){
            removedGroupID[0] = boardDevice[index + 1].groupID;
          }else{
            removedGroupID[0] = -1;
          }
        }
        else{
          targetGroupID[0] = -1;
          removedGroupID[0] = -1;
        }
  
        if (boardDevice[index-1].color == color){
          targetGroupID[1] = boardDevice[index-1].groupID;
        }else if(boardDevice[index - 1].color == enemyColor){
          if (boardDevice[index - 1].libertyNumber == 1){
            removedGroupID[1] = boardDevice[index - 1].groupID;
          }else{
            removedGroupID[1] = -1;
          }
        }
        else{
          targetGroupID[1] = -1;
          removedGroupID[1] = -1;
        }
        
        if (boardDevice[index+boardSize].color == color){
          targetGroupID[2] = boardDevice[index+boardSize].groupID;
        }else if(boardDevice[index + boardSize].color == enemyColor){
          if (boardDevice[index + boardSize].libertyNumber == 1){
            removedGroupID[0] = boardDevice[index + boardSize].groupID;
          }else{
            removedGroupID[2] = -1;
          }
        }
        else{
          targetGroupID[2] = -1;
          removedGroupID[2] = -1;
        }
  
        if (boardDevice[index-boardSize].color == color){
          targetGroupID[3] = boardDevice[index-boardSize].groupID;
        }else if(boardDevice[index - boardSize].color == enemyColor){
          if (boardDevice[index - boardSize].libertyNumber == 1){
            removedGroupID[0] = boardDevice[index - boardSize].groupID;
          }else{
            removedGroupID[3] = -1;
          }
        }
        else{
          targetGroupID[3] = -1;
          removedGroupID[3] = -1;
        }
  
    }
  
    globalLiberty[index] = 0;
    //hasStoneRemoved = false;
 
    __syncthreads();
  
    //@todo , check whether this fence is necessory.
    __threadfence_block();
  
  
    if (boardDevice[index].groupID == targetGroupID[0] ||
        boardDevice[index].groupID == targetGroupID[1] ||
        boardDevice[index].groupID == targetGroupID[2] ||
        boardDevice[index].groupID == targetGroupID[3] ){
      boardDevice[index].groupID = playPoint;
    }

   if (boardDevice[index].groupID == removedGroupID[0] ||
        boardDevice[index].groupID == removedGroupID[1] ||
        boardDevice[index].groupID == removedGroupID[2] ||
        boardDevice[index].groupID == removedGroupID[3] ){
      boardDevice[index].groupID = 0;
      boardDevice[index].color = GO_EMPTY;
      //hasStoneRemoved = true;
    }
   
 
    __syncthreads();
    __threadfence_block();
  
    updateStatus(boardDevice, index, color, globalLiberty, globalMoveValue, state);
  //
  //
  //
  //  if (boardDevice[index].pointGroup != NULL){
  //    debugFlagDevice[index].changeFlag = boardDevice[index].pointGroup.numberOfLiberty; 
  //    
  //  }
  //
  //
  //    debugFlagDevice[index].counter++;
  //  }
  //  
  
  }

__device__ void selectMove(BoardPoint *boardDevice, DebugFlag *debugFlagDevice, GoColor color, int *globalMoveValue, int *selectedMove){
  if (threadIdx.x == 0 && threadIdx.y == 0){
    int maxValue = -1;
    int maxIndex = 0;

    for (int i=0; i<totalSize; i++){
        if (globalMoveValue[i] > maxValue){
          maxValue = globalMoveValue[i];
          maxIndex = i;
        }
    }

    *selectedMove = maxIndex;
  }

}

  __global__
    void randomPlay(BoardPoint *boardDevice, DebugFlag *debugFlagDevice, GoColor color, hiprandState *state){
      int index = threadIdx.y*boardSize + threadIdx.x;

      __shared__ int globalLiberty[totalSize];
      __shared__ int globalMoveValue[totalSize];
      __shared__ int selectedMove;

      GoColor currentColor = invertColor(color);

      updateStatus(boardDevice, index, currentColor, globalLiberty, globalMoveValue, state);

      __syncthreads();
      __threadfence_block();
  
      selectMove(boardDevice, debugFlagDevice, currentColor, globalMoveValue, &selectedMove);

//#pragma unroll
      for (int i=0; i<500; i++){
 
        __syncthreads();
        __threadfence_block();
  
        if (selectedMove < 0){
          break;
        }

        playStone(boardDevice, debugFlagDevice, &selectedMove, currentColor, globalLiberty, globalMoveValue, state);
 
        currentColor = invertColor(currentColor);

        __syncthreads();
        __threadfence_block();
  
        selectMove(boardDevice, debugFlagDevice, currentColor, globalMoveValue, &selectedMove);
 

      }

    } 
 
  __global__
  void playBoard(BoardPoint *boardDevice, DebugFlag *debugFlagDevice, int row, int col, GoColor color, hiprandState *state){

    __shared__ int selectedMove;
    __shared__ int globalLiberty[totalSize]; // shared array to count the liberty of each group.
    __shared__ int globalMoveValue[totalSize]; 
 
    if (threadIdx.x == 0 && threadIdx.y ==0){
      // the corner point is special point for global operation.
        int playPoint = row*boardSize + col;
        selectedMove = playPoint;
    }

    __syncthreads();
    __threadfence_block();
    
    playStone(boardDevice, debugFlagDevice, &selectedMove, color, globalLiberty, globalMoveValue, state);
 
  
  }

//  __global__
//  void playBoard(BoardPoint *boardDevice, DebugFlag *debugFlagDevice, int row, int col, int color){
//    dim3 threadShape( boardSize, boardSize );
//    int numberOfBlock = 1;
//    playBoardInside<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, row, col, color);
//   
//  }
//   
 
//  __global__
//  void updateLegleMove(BoardPoint *boardDevice, DebugFlag *debugFlagDevice, int color){
//    int index = threadIdx.y*boardSize + threadIdx.x;
//  
//    if (boardDevice[index].color != GO_EMPTY){
//      boardDevice[index].isBlackLegal = false;
//      boardDevice[index].isWhiteLegal = false;
//    }else{
//      if (boardDevice[index - 1].color == GO_EMPTY ||
//          boardDevice[index + 1].color == GO_EMPTY ||
//          boardDevice[index - boardSize].color == GO_EMPTY ||
//          boardDevice[index + boardSize].color == GO_EMPTY){
//        boardDevice[index].isBlackLegal = true;
//        boardDevice[index].isWhiteLegal = true;
//
//      }else{
//        int totalLiberty = 0;
//        
//        if (boardDevice[index - 1].color == color){
//          totalLiberty = totalLiberty + boardDevice[index - 1].libertyNumber - 1;
//        }else if(boardDevice[index - 1].color == GO_EMPTY){
//          totalLiberty++;
//        }
//    
//        if (boardDevice[index + 1].color == color){
//          totalLiberty = totalLiberty + boardDevice[index + 1].libertyNumber - 1;
//        }else if(boardDevice[index + 1].color == GO_EMPTY){
//          totalLiberty++;
//        }
//    
//        if (boardDevice[index - boardSize].color == color){
//          totalLiberty = totalLiberty + boardDevice[index - boardSize].libertyNumber - 1;
//        }else if(boardDevice[index - boardSize].color == GO_EMPTY){
//          totalLiberty++;
//        }
//    
//        if (boardDevice[index + boardSize].color == color){
//          totalLiberty = totalLiberty + boardDevice[index + boardSize].libertyNumber - 1;
//        }else if(boardDevice[index + boardSize].color == GO_EMPTY){
//          totalLiberty++;
//        }
//    
//        debugFlagDevice[index].libertyCount = totalLiberty;
//    
//        if (totalLiberty > 0){
//          if (color == GO_BLACK){
//            boardDevice[index].isBlackLegal = true;
//          }else if (color == GO_WHITE){
//            boardDevice[index].isWhiteLegal = true;
//          }
//        }else{
//          if (color == GO_BLACK){
//            boardDevice[index].isBlackLegal = false;
//          }else if (color == GO_WHITE){
//            boardDevice[index].isWhiteLegal = false;
//          }
//     
//        }
//        
//      }       
// 
//   }// any of the points around boardDevice[index] is GO_EMPTY?
// }// boardDevice[index].color == GO_EMPTY?
//   

}//namespace


CUDABoard::CUDABoard(){
  this->currentPlayer = GO_BLACK;
  this->detailDebug = false;


  hipMalloc( (void**)&(this->boardDevice), this->valueSizeDevice );
  hipMalloc( (void**)&(this->debugFlagDevice), this->debugFlagSize );

  hipMalloc( (void**)&(this->stateDevice), valueSizeDevice  );

  dim3 threadShape( boardSize, boardSize );
  int numberOfBlock = 1;

  srand((unsigned int)time(NULL));
  
  initBoard<<<numberOfBlock, threadShape>>>(boardDevice, stateDevice, rand());
 
}

CUDABoard::~CUDABoard(){
  hipFree( boardDevice );
  hipFree( debugFlagDevice );
  hipFree( stateDevice );
   
}

void CUDABoard::Play(int row, int col, GoColor color){
//    GoPoint targetPoint = GoPointUtil::Pt(col, row);
//    Play(targetPoint, color);
  //dim3 threadShape( boardSize, boardSize  );
  dim3 threadShape( boardSize, boardSize );
  int numberOfBlock = 1;
  playBoard<<<numberOfBlock, threadShape>>>(this->boardDevice, this->debugFlagDevice, row, col, color, this->stateDevice);
}

void CUDABoard::Play(GoPoint p, GoColor color){

}

void CUDABoard::Play(GoPoint p){
  
}

void CUDABoard::RandomPlay(){
  dim3 threadShape( boardSize, boardSize );
  int numberOfBlock = 1;
  randomPlay<<<numberOfBlock, threadShape>>>(this->boardDevice, this->debugFlagDevice, this->currentPlayer, this->stateDevice);

  hipDeviceSynchronize();

}

void CUDABoard::RestoreData(){
  hipMemcpy( this->boardHost, this->boardDevice, this->valueSizeDevice, hipMemcpyDeviceToHost );
  hipMemcpy( this->debugFlagHost, this->debugFlagDevice, this->debugFlagSize, hipMemcpyDeviceToHost );

  hipDeviceSynchronize();


}

ostream& operator<<(ostream& out, const CUDABoard& cudaBoard){

  out << "Whole board:" << endl;


  for (int i=boardSize-1; i>=0; i--){
    for (int j=0; j<boardSize; j++){
      int index = i*boardSize + j;
      if (cudaBoard.boardHost[index].color == 0){
        out << ".";
      }else if (cudaBoard.boardHost[index].color == GO_BLACK){
        out << "o";
      }else if (cudaBoard.boardHost[index].color == GO_WHITE){
        out << "x";
      }else if (cudaBoard.boardHost[index].color == GO_BORDER){
        out << "H";
      }

       

    }
    if (cudaBoard.detailDebug){
      out << "     ";
      for (int j=0; j<boardSize; j++){
       int index = i*boardSize + j;
     
       if (cudaBoard.boardHost[index].color == GO_BORDER){
          out << "HHHH";
        }else {
          int value = cudaBoard.boardHost[index].moveValue%1000;
          std::stringstream ss;
          std::string outputString;
          //ss<<"      ";
          ss<< "___" << value;
          ss>>outputString;

          out << outputString.substr(outputString.length()-3);
          out << "|";

        }  
      } 
    }
    out << "\n";
   
  }

  return out;
}





//int main()
//{
//  
//  struct timeval start_tv;
//  gettimeofday(&start_tv,NULL);
//  
//  
// 
////  for (int i=0; i<19; i++){
////    playBoard<<<numberOfBlock, threadShape>>>(boardDevice, globalFlag, i, i, 2);
////  }
//
////  playBoard<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, 15, 12, 1);
//
//  //updateLegleMove<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, GO_BLACK);
//  //updateLegleMove<<<numberOfBlock, threadShape>>>(boardDevice, debugFlagDevice, GO_WHITE);
//
//  hipDeviceSynchronize();
//
//  hipMemcpy( boardHost, boardDevice, valueSizeDevice, hipMemcpyDeviceToHost );
//  hipMemcpy( debugFlagHost, debugFlagDevice, debugFlagSize, hipMemcpyDeviceToHost );
//
//
// 
//  hipDeviceSynchronize();
//
//  struct timeval end_tv;
//  gettimeofday(&end_tv,NULL);
// 
//  for (int i=boardSize-1; i>=0; i--){
//    for (int j=0; j<boardSize; j++){
//      int index = i*boardSize + j;
//      if (boardHost[index].color == 0){
//        printf(".");
//      }else if (boardHost[index].color == GO_BLACK){
//        printf("o");
//      }else if (boardHost[index].color == GO_WHITE){
//        printf("x");
//      }else if (boardHost[index].color == GO_BORDER){
//        printf("H");
//      }
//    }
//    printf("\n");
//   
//  }
//
////  for (int i=boardSize-1; i>=0; i--){
////    for (int j=0; j<boardSize; j++){
////      int index = i*boardSize + j;
//////      if (boardHost[index].color == GO_BLACK || boardHost[index].color == GO_WHITE){
////        printf("%d, %d | ", boardHost[index].groupID, boardHost[index].libertyNumber);
//////      } else if (boardHost[index].color == GO_EMPTY) {
//////        printf("   ,   | ");
//////      }
////    }
////    printf("\n");
////   
////  }
//
//  for (int i=boardSize-1; i>=0; i--){
//    for (int j=0; j<boardSize; j++){
//      int index = i*boardSize + j;
//      if (boardHost[index].color == GO_BORDER){
//        printf("H");
//      }else{
//        if (boardHost[index].isBlackLegal){
//          printf("o");
//        }else {
//          printf(".");
//        }
//      }
//    }
//
//    printf("        ");
//
//    for (int j=0; j<boardSize; j++){
//      int index = i*boardSize + j;
//      if (boardHost[index].color == GO_BORDER){
//        printf("H");
//      }else{
//        if (boardHost[index].isWhiteLegal){
//          printf("x");
//        }else {
//          printf(".");
//        }
//      }
//    }
//    
//    printf("\n");
//   
//  }
//
//
//
////  for (int i=boardSize-1; i>=0; i--){
////    for (int j=0; j<boardSize; j++){
////      int index = i*boardSize + j;
////      printf("%d | ", debugFlagHost[index].libertyCount);
////      }
////    printf("\n");
////   
////  }
//
//
//  printf("\n");
//
//  if(end_tv.tv_usec >= start_tv.tv_usec){
//    printf("time %lu:%lu\n",end_tv.tv_sec - start_tv.tv_sec,  end_tv.tv_usec - start_tv.tv_usec);
//  }else{
//    printf("time %lu:%lu\n",end_tv.tv_sec - start_tv.tv_sec - 1,  1000000 - start_tv.tv_usec + end_tv.tv_usec);
//  }
//
//  
//  return EXIT_SUCCESS;
//  
//}
